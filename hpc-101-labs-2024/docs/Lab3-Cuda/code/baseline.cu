#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <cstdlib>
#include <iomanip>
#include <iostream>
#include <random>
#include <omp.h>

const int block_size = 16;
const int size = 10001;     // Matrix Size (size * size)
const int iter = 2;     // Number of iterations

#define a(_x, _y) a[(_x) * size + (_y)]
#define b(_x, _y) b[(_x) * size + (_y)]
#define result(_x, _y) result[(_x) * size + (_y)]
#define CUDA_CALL(func)                                               \
  {                                                                   \
    hipError_t e = (func);                                           \
    if (!(e == hipSuccess || e == hipErrorDeinitialized))         \
    {                                                                 \
      fprintf(stderr, "CUDA: %s:%d: error: %s\n", __FILE__, __LINE__, \
              hipGetErrorString(e));                                 \
      abort();                                                        \
    }                                                                 \
  }
#define CUBLAS_CALL(func)                                             \
  {                                                                   \
    hipblasStatus_t e = (func);                                        \
    if (!(e == HIPBLAS_STATUS_SUCCESS))                                \
    {                                                                 \
      fprintf(stderr, "CUBLAS: %s:%d: error: %d\n", __FILE__, __LINE__, \
              e);                                 \
      abort();                                                        \
    }                                                                 \
  }

/// \brief Simply generate a random matrix.
void Generate(double *const a) {
  srand(time(NULL));
  // Matrix row.
#pragma omp parallel for
  for (int i = 0; i < size; ++i) {
    // Matrix column.
    for (int j = 0; j < size; ++j) {
      // Matrix element.
      a(i, j) = rand() % 100 / 100.0f;
    }
  }
}

/// \brief Check the correctness of the result and compare performace by using Cublas.
void CublasImplete(const double *__restrict__ a,
                   const double *__restrict__ b,
                   double *__restrict__ result,
                   hipEvent_t *start_cublas, hipEvent_t *stop_cublas) {
  double *a_kernel_1, *a_kernel_2, *b_kernel, *result_kernel;
  CUDA_CALL(hipMalloc(&a_kernel_1, size * size * sizeof(double)));
  CUDA_CALL(hipMemcpy(a_kernel_1, a, size * size * sizeof(double), hipMemcpyHostToDevice));
  CUDA_CALL(hipMalloc(&a_kernel_2, size * size * sizeof(double)));
  CUDA_CALL(hipMemcpy(a_kernel_2, a_kernel_1, size * size * sizeof(double), hipMemcpyDeviceToDevice));
  CUDA_CALL(hipMalloc(&b_kernel, size * size * sizeof(double)));
  CUDA_CALL(hipMemcpy(b_kernel, b, size * size * sizeof(double), hipMemcpyHostToDevice));
  CUDA_CALL(hipMalloc(&result_kernel, size * size * sizeof(double)));

  hipEventRecord(*start_cublas);
  // Use cublasDgeam to (A + (k - 1) * B) + B -> A + k * B.
  hipblasHandle_t handle;
  CUBLAS_CALL(hipblasCreate(&handle));
  double alpha = 1.0f;
  double betageam = 1.0f;
  double betagemm = 0.0f;
  for (int i = 0; i < iter; ++i) {
    CUBLAS_CALL(hipblasDgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, size, size, &alpha, a_kernel_2, size, &betageam, b_kernel, size, a_kernel_2, size));
    
    CUBLAS_CALL(hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, size, size, size, &alpha, a_kernel_2, size, a_kernel_1, size, &betagemm, result_kernel, size));
    double *tmp = a_kernel_1;
    a_kernel_1 = result_kernel;
    result_kernel = tmp;
  }

  hipEventRecord(*stop_cublas);
  hipEventSynchronize(*stop_cublas);

  CUBLAS_CALL(hipblasGetMatrix(size, size, sizeof(double), a_kernel_1, size, result, size));
  CUBLAS_CALL(hipblasDestroy(handle));

  return;
}

/// \brief Check the correctness of the result.
void Verify(const double *const result,
            const double *const result_cublas) {
  bool correct = true;
  for (int i = 0; i < size * size; ++i) {
    // check if there is inf
    if (result[i] == INFINITY || result[i] == -INFINITY) {
      std::cout << "\x1b[31m"
                   "Wrong Answer"
                   "\x1b[0m"
                   " at ("
                << i / size << ", " << i % size << "): ";
      std::cout << "expected " << std::setprecision(6) << result_cublas[i]
                << ", got " << result[i]
                << std::endl;
      correct = false;
      break;
    }
    if (fabs(result[i]) < 1e-6 || fabs(result_cublas[i]) < 1e-6) {
      std::cout << "\x1b[31m"
                   "Wrong Answer"
                   "\x1b[0m"
                   " at ("
                << i / size << ", " << i % size << "): ";
      std::cout << "expected " << std::setprecision(6) << result_cublas[i]
                << ", got " << result[i]
                << std::endl;
      correct = false;
      break;
    }
    double error = fabs(result[i] - result_cublas[i]) / result_cublas[i];
    if (error > 1e-6) {
      correct = false;
      std::cout << "\x1b[31m"
                   "Wrong Answer"
                   "\x1b[0m"
                   " at ("
                << i / size << ", " << i % size << "): ";
      std::cout << "expected " << std::setprecision(6) << result_cublas[i]
                << ", got " << result[i]
                << std::endl;
      break;
    }
  }
  if (correct) {
    std::cout << "\x1b[32m"
                "Correct"
                "\x1b[0m"
              << std::endl;
  }
  return;
}

/// \brief Let A to be A + B.
__global__ void AdderCudaKernel(double *__restrict__ a,
                                  const double *__restrict__ b)
{
  const int i = blockIdx.x * block_size + threadIdx.x;
  const int j = blockIdx.y * block_size + threadIdx.y;
  if (i < size && j < size) {
    a(i, j) += b(i, j);
  }
}

/// \brief Do Matrix Multiplication on GPU.
__global__ void MultipleCudaKernel(const double *__restrict__ a, 
                                     const double *__restrict__ b, 
                                     double *__restrict__ result) 
{
  const int i = blockIdx.x * block_size + threadIdx.x;
  const int j = blockIdx.y * block_size + threadIdx.y;
  if (i < size && j < size) {
    result(i, j) = 0;
    for (int k = 0; k < size; ++k) {
      result(i, j) += a(i, k) * b(k, j);
    }
  }
}

// Naive implementation, only for testing correctness and precision
void MultipleCuda(const double *const a, const double *const b, double *const result,
                   hipEvent_t *start_e, hipEvent_t *stop_e) 
{
  double *a_kernel, *b_kernel, *copy_kernel, *result_kernel;
  CUDA_CALL(hipMalloc(&a_kernel, size * size * sizeof(double)));
  CUDA_CALL(hipMemcpy(a_kernel, a, size * size * sizeof(double), hipMemcpyHostToDevice));
  CUDA_CALL(hipMalloc(&b_kernel, size * size * sizeof(double)));
  CUDA_CALL(hipMemcpy(b_kernel, b, size * size * sizeof(double), hipMemcpyHostToDevice));
  CUDA_CALL(hipMalloc(&copy_kernel, size * size * sizeof(double)));
  CUDA_CALL(hipMemcpy(copy_kernel, a_kernel, size * size * sizeof(double), hipMemcpyDeviceToDevice));
  CUDA_CALL(hipMalloc(&result_kernel, size * size * sizeof(double)));
  
  // Start Timer.
  hipEventRecord(*start_e);
  
  // Run Matrix Multiplication.
  // Parameters to be set:
  dim3 grid((size + block_size - 1) / block_size,
            (size + block_size - 1) / block_size);
  dim3 block(block_size, block_size);

  // Calculate \Prod_{k=0}^{n} (A + k * B).
  for (int i = 0; i < iter; ++i) {
    // @note: you can also use CUDA API to launch a cuda kernel function,
    // __host__ cudaError_t cudaLaunchKernel;
    // Perform (A + (k - 1) * B) + B -> A + k * B.
    AdderCudaKernel<<<grid, block>>>(copy_kernel, b_kernel);
    CUDA_CALL(hipDeviceSynchronize());
    // Perform A * B -> Result.
    MultipleCudaKernel<<<grid, block>>>(a_kernel, copy_kernel, result_kernel);
    CUDA_CALL(hipDeviceSynchronize());

    // Swap pointers between A and Result.
    double *tmp = a_kernel;
    a_kernel = result_kernel;
    result_kernel = tmp;
  }

  // Stop Timer
  hipEventRecord(*stop_e);
  hipEventSynchronize(*stop_e);

  // At the end of the loop, the result is in a_kernel.
  CUDA_CALL(hipMemcpy(result, a_kernel, size * size * sizeof(double), hipMemcpyDeviceToHost));
  hipFree(a_kernel);
  hipFree(b_kernel);
  hipFree(copy_kernel);
  hipFree(result_kernel);
}

int main() {
  auto a = new double[size * size];
  auto b = new double[size * size];
  auto result = new double[size * size];
  auto result_cublas = new double[size * size];
  std::cout << "Generating input matrices... \n";
  Generate(a);
  Generate(b);

  hipEvent_t start_e, stop_e;
  hipEventCreate(&start_e);
  hipEventCreate(&stop_e);

  // Perform Matrix Multiplication on GPU.
  std::cout << "Custom Matrix Multiplication on GPU... \n";
  MultipleCuda(a, b, result, &start_e, &stop_e);

  hipEvent_t start_cublas, stop_cublas;
  hipEventCreate(&start_cublas);
  hipEventCreate(&stop_cublas);
  std::cout << "cuBLAS Matrix Multiplication on GPU... \n";
  CublasImplete(a, b, result_cublas, &start_cublas, &stop_cublas);

  std::cout << "Verifying... \n";
  // Verify the result.
  Verify(result, result_cublas);

  // Calculate to evaluate performance.
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start_e, stop_e);
  std::cout << "Custom: " << milliseconds << " milliseconds" << std::endl;
  hipEventElapsedTime(&milliseconds, start_cublas, stop_cublas);
  std::cout << "cuBLAS: " << milliseconds << " milliseconds" << std::endl;
  hipEventDestroy(start_e);
  hipEventDestroy(stop_e);
  hipEventDestroy(start_cublas);
  hipEventDestroy(stop_cublas);

  // Delete allocated memory.
  delete[] a;
  delete[] b;
  delete[] result;
  return 0;
}